#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>

#define MASK_WIDTH 3
#define TILE_WIDTH 16
#define CHANNELS 3
#define ITERATIONS 100 

_constant_ float d_mask[MASK_WIDTH][MASK_WIDTH] = {
    {1, 2, 1},
    {2, 4, 2},
    {1, 2, 1}
};

_global_ void gaussianBlur(unsigned char *inputImage, unsigned char *outputImage, int width, int height) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_o = blockIdx.y * blockDim.y + ty;
    int col_o = blockIdx.x * blockDim.x + tx;

    float result[CHANNELS] = {0};

    if (row_o < height && col_o < width) {
        for (int i = -MASK_WIDTH/2; i <= MASK_WIDTH/2; ++i) {
            for (int j = -MASK_WIDTH/2; j <= MASK_WIDTH/2; ++j) {
                int row_i = min(max(row_o + i, 0), height - 1);
                int col_i = min(max(col_o + j, 0), width - 1);

                for (int c = 0; c < CHANNELS; ++c) {
                    result[c] += inputImage[(row_i * width + col_i) * CHANNELS + c] * d_mask[i + MASK_WIDTH/2][j + MASK_WIDTH/2];
                }
            }
        }

        for (int c = 0; c < CHANNELS; ++c) {
            outputImage[(row_o * width + col_o) * CHANNELS + c] = (unsigned char)(result[c] / 16.0f);
        }
    }
}

int main() {
    
    cv::Mat inputImage = cv::imread("Dude.jpg", cv::IMREAD_COLOR);

    if (inputImage.empty()) {
        printf("Error: Unable to load image.\n");
        return -1;
    }

    int width = inputImage.cols;
    int height = inputImage.rows;

    // Allocate memory for input and output images on GPU
    size_t imageSize = width * height * CHANNELS * sizeof(unsigned char);
    unsigned char *d_inputImage, *d_outputImage;
    hipMalloc(&d_inputImage, imageSize);
    hipMalloc(&d_outputImage, imageSize);

    // Copy input image to GPU memory
    hipMemcpy(d_inputImage, inputImage.data, imageSize, hipMemcpyHostToDevice);

    // Define grid and block dimensions for CUDA kernel
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 numBlocks((width + TILE_WIDTH - 1) / TILE_WIDTH, (height + TILE_WIDTH - 1) / TILE_WIDTH);

    // Apply multiple iterations of Gaussian blur
    for (int i = 0; i < ITERATIONS; ++i) {
        gaussianBlur<<<numBlocks, threadsPerBlock>>>(d_inputImage, d_outputImage, width, height);
        std::swap(d_inputImage, d_outputImage); // Swap input and output for next iteration
    }

    // Copy final output image from GPU to CPU
    hipMemcpy(inputImage.data, d_inputImage, imageSize, hipMemcpyDeviceToHost);

    // Save the output image
    cv::imwrite("Dude_blurred.jpg", inputImage);

    // Free GPU memory
    hipFree(d_inputImage);
    hipFree(d_outputImage);

    return 0;
}